
#include <hip/hip_runtime.h>
#include <stdio.h>

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numCRows && col < numCColumns) {
        float sum = 0.0f;
        for (int i = 0; i < numAColumns; ++i) {
            sum += A[row * numAColumns + i] * B[i * numBColumns + col];
        }
        C[row * numCColumns + col] = sum;
    }
}

int main(int argc, char **argv) {
    float *hostA; // The A matrix
    float *hostB; // The B matrix
    float *hostC; // The output C matrix
    float *deviceA;
    float *deviceB;
    float *deviceC;
    int numARows = 1024;    // number of rows in matrix A
    int numAColumns = 1024; // number of columns in matrix A
    int numBRows = 1024;    // number of rows in matrix B
    int numBColumns = 1024; // number of columns in matrix B
    int numCRows = numARows; // number of rows in matrix C
    int numCColumns = numBColumns; // number of columns in matrix C

    hostA = (float *)malloc(numARows * numAColumns * sizeof(float));
    hostB = (float *)malloc(numBRows * numBColumns * sizeof(float));
    hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));

    // Allocate the hostC matrix
    // Allocate GPU memory here
    hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
    hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
    hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));

    // Copy memory to the GPU here
    hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);

    // Initialize the grid and block dimensions here
    dim3 blockDim(16, 16);
    dim3 gridDim((numCColumns + blockDim.x - 1) / blockDim.x, (numCRows + blockDim.y - 1) / blockDim.y);

    // Launch the GPU Kernel here
    matrixMultiply<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    hipDeviceSynchronize();

    // Copy the GPU memory back to the CPU here
    hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);

    // Free the GPU memory here
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
